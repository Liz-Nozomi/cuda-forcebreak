#include "hip/hip_runtime.h"


#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>
#include <fstream>
#include "c_utils.h"
#include "des.h"
#include "des_utils.h"
#include "bit_utils.h"
#include "des_consts.h"
#include "des_kernel.h"
#include "cuda_utils.h"


static void CheckCudaErrorAux(const char *, unsigned, const char *,
    hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux(const char *file, unsigned line,
    const char *statement, hipError_t err) {
  if (err == hipSuccess)
    return;
  std::cerr << statement << " returned " << hipGetErrorString(err) << "("
      << err << ") at " << file << ":" << line << std::endl;
  exit(1);
}

__global__ void kernel(int* resultsDevice, int dim, uint64_t* hashesDevice) {
	int mI = threadIdx.y+blockIdx.y*blockDim.y;
	int yI = threadIdx.x+blockIdx.x*blockDim.x + 1940;
	int dI = threadIdx.z;
	uint64_t key = yI*10000+mI*100+dI;
	uint64_t encoded = 0;
	encoded = full_des_encode_block(key, key);
	for(int i=0;i<dim;i++){
		if (hashesDevice[i] == encoded){
			resultsDevice[i] = 1;
		}
	}
}


int main(void){
	#define dim 1000
	int resultsHost[dim];
	FILE * fp;
	char * line = NULL;
	size_t len = 0;
	ssize_t read;
	uint64_t hashesHost[dim];
	int k=0;
	fp = fopen("PswDb/db1000.txt", "r");
	while ((read = getline(&line, &len, fp)) != -1) {
		char* hash =(char*) malloc(sizeof(char)*9);
		for(int i = 0; i<9; i++){
		  hash[i]=line[i];
		}
		hash[8]= '\0'; //string termination
		hashesHost[k]=full_des_encode_block(atoi(hash),atoi(hash));
		k++;
	}
	fclose(fp);
	free(line);

	//GPU memory allocation
	uint64_t* hashesDevice;
	int* resultsDevice;

	CUDA_CHECK_RETURN( hipMalloc((void **)&hashesDevice, dim * sizeof(uint64_t)) );

	CUDA_CHECK_RETURN( hipMemcpy(hashesDevice, hashesHost, dim * sizeof(uint64_t), hipMemcpyHostToDevice) );

	CUDA_CHECK_RETURN( hipMalloc((void **) &resultsDevice, sizeof(int) * dim));

	dim3 dimGrid(8,5);
	dim3 dimBlock(10,3,32);//
	clock_t start = clock();
	kernel<<<dimGrid,dimBlock>>>(resultsDevice,dim,hashesDevice);
	// copy results from device memory to host
	hipDeviceSynchronize();
	CUDA_CHECK_RETURN(
	  hipMemcpy(resultsHost, resultsDevice, dim * sizeof(int),
		  hipMemcpyDeviceToHost));
	clock_t end = clock();
	float seconds = (float) (end - start) / CLOCKS_PER_SEC;
	hipFree(hashesDevice);
	hipFree(resultsDevice);

	int count = 0;
	for(int i = 0; i < dim; i++){
		if(resultsHost[i] == 1){
			count++;
		}
	}
	printf("found hashes: %d\n", count);
	printf("time: %f",seconds);
	return 0;
}
